#include "hip/hip_runtime.h"
#include "cudalbm.h"
#include "seriallbm.hpp"

inline void chkerr(hipError_t code) {
  if (code != hipSuccess) {
    std::cerr << "ERROR!!!:" << hipGetErrorString(code) << std::endl;
    exit(-1);
  }
}

void accuracyTest(double *ux, double *uy, double *uxd, double *uyd, int sz) {

  double error{0};
  double tol{1e-4};
  bool success = true;

  for (int i = 0; i < sz; i++) {

    if (abs(ux[i] - uxd[i]) > tol) {
      std::cout << "Outputs don't match at i = \t" << i << "\n";
      success = false;
      break;
    }

    if (abs(uy[i] - uyd[i]) > tol) {
      std::cout << "Outputs don't match at i = \t" << i << "\n";
      success = false;
      break;
    }
  }

  if (success)
    std::cout << "SUCCESS, Outputs match \n";
}

int main() {

  int szf = Ny * Nx * Q9;
  int sz = Nx * Ny;

  constexpr double tau = 1;
  // double g = 0.0001373;
  // double U = 0.0333*1.5;

  constexpr double g = 0.001102;
  constexpr double U = 0.1;

  double *fvals = new double[szf];
  std::fill(fvals, fvals + szf, 0);

  double *fvalsinit = new double[szf];
  std::fill(fvalsinit, fvalsinit + szf, 0);

  double *rho = new double[sz];
  std::fill(rho, rho + sz, 1);

  double *ux = new double[sz];
  std::fill(ux, ux + sz, 0);

  double *uy = new double[sz];
  std::fill(uy, uy + sz, 0);

  double *rhoprev = new double[sz];
  std::fill(rhoprev, rhoprev + sz, 0);

  double *uxprev = new double[sz];
  std::fill(uxprev, uxprev + sz, 0);

  double *uyprev = new double[sz];
  std::fill(uyprev, uyprev + sz, 0);

  double *fvalsprev = new double[szf];
  std::fill(fvalsprev, fvalsprev + szf, 0);

  double *feq = new double[Q9];
  std::fill(feq, feq + Q9, 0);

  setInitialVelocity(ux, uy, U);

  calcEqDis(fvalsinit, rho, ux, uy, g, tau);
  calcEqDis(fvalsprev, rho, ux, uy, g, tau);
  calcEqDis(fvals, rho, ux, uy, g, tau);

  double *ex = new double[Q9]{0, 1, 0, -1, 0, 1, -1, -1, 1};
  double *ey = new double[Q9]{0, 0, 1, 0, -1, 1, 1, -1, -1};

  double c = 1;
  int Niter = 1;
  double tol = 1e-8;

  hipEvent_t seq_start, seq_stop;
  float seq_time;
  hipEventCreate(&seq_start);
  hipEventCreate(&seq_stop);
  hipEventRecord(seq_start);

  // performLBMPushOut( fvals, rho, ux, uy, ex, ey, g, tau, szf, Niter );
  performLBMPullIn(fvals, fvalsprev, feq, rho, ux, uy, uxprev, uyprev, ex, ey,
                   g, tau, szf, Niter, tol);

  hipEventRecord(seq_stop);
  hipEventSynchronize(seq_stop);
  hipEventElapsedTime(&seq_time, seq_start, seq_stop);

  calcMacroscopic(fvals, rho, ux, uy, ex, ey);

  double *dfvals, *dfvalsprev, *dex, *dey;
  chkerr(hipMalloc((void **)&dfvals, sizeof(double) * szf));
  chkerr(hipMalloc((void **)&dfvalsprev, sizeof(double) * szf));
  chkerr(hipMalloc((void **)&dex, sizeof(double) * Q9));
  chkerr(hipMalloc((void **)&dey, sizeof(double) * Q9));

  chkerr(hipMemcpy(dfvals, fvalsinit, sizeof(double) * szf,
                    hipMemcpyHostToDevice));
  chkerr(hipMemcpy(dfvalsprev, fvalsinit, sizeof(double) * szf,
                    hipMemcpyHostToDevice));
  chkerr(hipMemcpy(dex, ex, sizeof(double) * Q9, hipMemcpyHostToDevice));
  chkerr(hipMemcpy(dey, ey, sizeof(double) * Q9, hipMemcpyHostToDevice));

  double *rhod = new double[sz];
  std::fill(rhod, rhod + sz, 0);

  double *uxd = new double[sz];
  std::fill(uxd, uxd + sz, 0);

  double *uyd = new double[sz];
  std::fill(uyd, uyd + sz, 0);

  dim3 block_spec;
  block_spec.x = BLOCKSIZE;

  int gsize = ceil(sz / ((double)BLOCKSIZE));

  dim3 grid_spec(gsize, 1);

  int t = 0;
  double error{1e9};
  std::cout << "Entering Device Code \n";

  hipEvent_t par_start, par_stop;
  float par_time;
  hipEventCreate(&par_start);
  hipEventCreate(&par_stop);
  hipEventRecord(par_start);

  while (t < Niter) {
    parlbm<<<grid_spec, block_spec>>>(dfvals, dfvalsprev, dex, dey, g, tau,
                                      szf);
    t++;

    if (t < Niter)
      std::swap(dfvals, dfvalsprev);
  }

  hipEventRecord(par_stop);
  hipEventSynchronize(par_stop);
  hipEventElapsedTime(&par_time, par_start, par_stop);

  hipError_t err = hipGetLastError();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  double *fvalsd = new double[szf];
  std::fill(fvalsd, fvalsd + szf, 0);

  chkerr(
      hipMemcpy(fvalsd, dfvals, sizeof(double) * szf, hipMemcpyDeviceToHost));

  chkerr(hipFree(dfvals));
  chkerr(hipFree(dfvalsprev));
  chkerr(hipFree(dex));
  chkerr(hipFree(dey));

  calcMacroscopic(fvalsd, rhod, uxd, uyd, ex, ey);

  accuracyTest(ux, uy, uxd, uyd, sz);

  printu(ux, uy, "velocity.txt");
  printval(rho, "rho.txt");
  printf(fvals, "fvals.txt");

  printu(uxd, uyd, "velocitydevice.txt");
  printval(rhod, "rhodevice.txt");
  printf(fvalsd, "fvalsdevice.txt");

  // std::string filenameval = "timecalcNx=" + std::to_string(Nx) + "Ny=" + std::to_string(Ny) + ".txt";

  std::ofstream fileval( "timecalc.txt" );

  fileval << seq_time << "\n";
  fileval << par_time << "\n";
  fileval << seq_time/par_time << "\n";

  double seqmlups = sz*Niter*1.0/std::pow( 10, 6 )/seq_time;
  double parmlups = sz*Niter*1.0/std::pow( 10, 6 )/par_time;

  // fileval << seqmlups << "\n";
  // fileval << parmlups << "\n";
  // fileval << sz << "\n";

  return 0;
}